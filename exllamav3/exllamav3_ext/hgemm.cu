#include "hip/hip_runtime.h"
#include "hgemm.cuh"
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_fp16.h>
#include "util.h"
#include "util.cuh"

/*

Row-major float16 matmul using cuBLAS, a @ b -> c

*/

void hgemm
(
    at::Tensor a,
    at::Tensor b,
    at::Tensor c
)
{
    const at::cuda::OptionalCUDAGuard device_guard(a.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    TORCH_CHECK_DTYPE(a, kHalf);
    TORCH_CHECK_DTYPE(b, kHalf);
    TORCH_CHECK_DTYPE(c, kHalf);
    TORCH_CHECK_DIM(a, 2);
    TORCH_CHECK_DIM(b, 2);
    TORCH_CHECK_DIM(c, 2);
    TORCH_CHECK_SHAPES(a, 0, c, 0, 1);
    TORCH_CHECK_SHAPES(a, 1, b, 0, 1);
    TORCH_CHECK_SHAPES(b, 1, c, 1, 1);

    const half* a_ptr = (const half*) a.data_ptr();
    const half* b_ptr = (const half*) b.data_ptr();
    half* c_ptr = (half*) c.data_ptr();

    int size_m = a.size(0);
    int size_k = a.size(1);
    int size_n = b.size(1);

    hipblasHandle_t cublas_handle = at::cuda::getCurrentCUDABlasHandle();

    half alpha_ = __float2half(1.0f);
    half beta_ = __float2half(0.0f);
    hipblasSetStream(cublas_handle, stream);
    hipblasHgemm
    (
        cublas_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        size_n, size_m, size_k,
        &alpha_, b_ptr, size_n,
                 a_ptr, size_k,
        &beta_,  c_ptr, size_n
    );
}
